#include <stdio.h>
#include <wb.h>
#include "Otsus_Method.h"
#include "filters.h"
#include "non_max_supp.h"
#include "Edge_Connection.h"

#define FILTERSIZE 3

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

int main(int argc, char *argv[])
{

	//////////////////////////////
	// Parameter Initialization //
	//////////////////////////////


	// Image parameters for wbLib
	wbArg_t args;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	wbImage_t inputImage;
	wbImage_t outputImage;

	// Host side parameters
	float *hostInputImageData;
	float *hostGrayImageData; 
	float *hostBlurImageData;
	float *hostGradMagData;
	float *hostGradPhaseData;
	//float *hostEdgeImage;
	//float *hostWeakEdgeImage;

	// Device side parameters
	float *deviceInputImageData;
	float *deviceGrayImageData;
	float *deviceBlurImageData;
	float *deviceGradMagData;
	float *deviceGradPhaseData;
	//float *deviceEdgeImage;
	//float *deviceWeakEdgeImage;
	
	// Filtering parameters
	float *BlurImageData;
	float *GradMagData;
	float *GradPhaseData;
	float *NmsImageData;

	// Otsu's Method parameters
	unsigned int *histogram;


	////////////////////
	// Image Handling //
	////////////////////


	// Parse the input arguments
	args = wbArg_read(argc, argv);

	// Read input file
	inputImageFile = wbArg_getInputFile(args, 0);

	// Import input image 
	inputImage = wbImport(inputImageFile);

	// Scrape info from input image
	imageWidth  = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage);
	
	// Define new output image
	outputImage = wbImage_new(imageWidth, imageHeight, 1);

	// Define output image data
	hostInputImageData = wbImage_getData(inputImage);

	// CHANGE THIS TO CHANGE OUTPUT IMAGE
	BlurImageData = wbImage_getData(outputImage);


	////////////////////////////////
	// Host Memory Initialization //
	////////////////////////////////


	// Allocate memory on host
	hostGrayImageData     = (float *)malloc(imageHeight*imageWidth*sizeof(float));
	hostBlurImageData     = (float *)malloc(imageHeight*imageWidth*sizeof(float));
	hostGradMagData 	  = (float *)malloc(imageHeight*imageWidth*sizeof(float));
	hostGradPhaseData 	  = (float *)malloc(imageHeight*imageWidth*sizeof(float));

	// Allocate memory for serial filtering and initialize to 0
	//BlurImageData     = (float *)calloc(imageHeight*imageWidth, sizeof(float));
	GradMagData    		= (float *)calloc(imageHeight*imageWidth, sizeof(float));
	GradPhaseData 		= (float *)calloc(imageHeight*imageWidth, sizeof(float));
	NmsImageData      	= (float *)calloc(imageHeight*imageWidth, sizeof(float));

	// Allocate memory on host and initialize to 0
	histogram = (unsigned int *)calloc(256, sizeof(unsigned int));


	/////////////////////////
	// Image Preprocessing //
	/////////////////////////


	// Create filter skeleton
	double filter[FILTERSIZE][FILTERSIZE];

	// Fill the gaussian filter
	populate_blur_filter(filter);

	// ?????
	int filterSize = (int)FILTERSIZE;


	//////////////////////////////////
	// Device Memory Initialization //
	//////////////////////////////////


	// Start total program timer
	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	// Start memory allocation timer
	wbTime_start(GPU, "Doing GPU memory allocation");

	// Allocate memory on device
	hipMalloc((void **)&deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **)&deviceGrayImageData, imageWidth*imageHeight*sizeof(float));
	hipMalloc((void **)&deviceBlurImageData, imageWidth*imageHeight*sizeof(int));
	hipMalloc((void **)&deviceGradMagData, imageWidth*imageHeight*sizeof(int));
	hipMalloc((void **)&deviceGradPhaseData, imageWidth*imageHeight*sizeof(int));
	//hipMalloc((void **)&deviceEdgeImage, imageWidth*imageHeight*sizeof(float));
	//hipMalloc((void **)&deviceWeakEdgeImage, imageWidth*imageHeight*sizeof(float))

	// Stop memory allocation timer
	wbTime_stop(GPU, "Doing GPU memory allocation");

	// Start memory copy timer
	wbTime_start(Copy, "Copying data to the GPU");

	// Copy input image from host to device
	hipMemcpy(deviceInputImageData, hostInputImageData, imageChannels*imageWidth*imageHeight*sizeof(float), hipMemcpyHostToDevice);


	///////////////////
	// GPU Execution //
	///////////////////


	// Start computation timer
	wbTime_start(Compute, "Doing the computation on the GPU");

	// Number of threads/block is 16
	int blocksize = 16;

	// Initialize x and y block dimension to blocksize
	dim3 BlockDim(blocksize,blocksize);

	// Set x and y grid dimension 
	dim3 GridDim(((imageWidth+BlockDim.x-1)/BlockDim.x), ((imageHeight+BlockDim.y-1)/BlockDim.y));  

	// Call RGB to grayscale conversion kernel
	ColorToGrayscale<<<GridDim, BlockDim>>>(deviceInputImageData, deviceGrayImageData, imageWidth, imageHeight);

	// Call image burring kernel
	//Conv2D<<<GridDim, BlockDim>>>(deviceGrayImageData, deviceBlurImageData, filter, imageWidth, imageHeight, filterSize);

	// Call sobel filtering kernel
	//GradientSobel<<<GridDim, BlockDim>>>(deviceBlurImageData, deviceSobelImageData, deviceSobelImageData, imageHeight, imageWidth); 

	// Call strong weak and non edge detection kernel
	//thresh_detection_global_kernel<<<GridDim, BlockDim>>>(image, deviceWeakEdgeImage, deviceEdgeImage, deviceThresh, imageHeight, imageWidth);

	//Call edge detection kernel
	//edge_connection_global_kernel<<<GridDim, BlockDim>>>(deviceWeakEdgeImage, deviceEdgeImage, imageHeight, imageWidth);

	// Stop computation timer
	wbTime_stop(Compute, "Doing the computation on the GPU");


	////////////////////
	// Device Results //
	////////////////////


	// Start device memory copy timer
	wbTime_start(Copy, "Copying data from the GPU");

	// Copy data from device back to host
	hipMemcpy(hostGrayImageData, deviceGrayImageData, imageWidth*imageHeight*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(hostBlurImageData, deviceBlurImageData, imageWidth*imageHeight*sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(hostGrayImageData, deviceGrayImageData, imageWidth*imageHeight*sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(hostSobelImageData, deviceSobelImageData, imageWidth*imageHeight*sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(hostGradientImageData, deviceGradientImageData, imageWidth*imageHeight*sizeof(float), hipMemcpyHostToDevice); 

	// Stop memory timer
	wbTime_stop(Copy, "Copying data from the GPU");

	// Stop total program timer
	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	
	////////////////////
	// Host Execution //
	////////////////////

	// Blur image using Gaussian Kernel
	Conv2DSerial(hostGrayImageData, BlurImageData, filter, imageWidth, imageHeight, filterSize);

	// Calculate gradient using Sobel Operators
	GradientSobelSerial(BlurImageData, GradMagData, GradPhaseData, imageHeight, imageWidth);

  	// Suppress non-maximum pixels along gradient
  	//nms(SobelImageData, NmsImageData, GradientImageData, imageHeight, imageWidth);

	// Calculate histogram of blurred image
	Histogram_Sequential(BlurImageData, histogram, imageWidth, imageHeight);

	// Calculate threshold using Otsu's Method
	double thresh = Otsu_Sequential(histogram);

	


	////////////////////
	// Debugging Info //
	////////////////////


	// Print info
	printf("\n");
	printf("Width = %u\n",imageWidth);
	printf("Height = %u\n",imageHeight);
	printf("InputImage[0] = %f\n",hostInputImageData[0]);
	printf("Histogram[0] = %u\n",histogram[0]);
	printf("Histogram[1] = %u\n",histogram[1]);
	printf("Histogram[20] = %u\n",histogram[20]);
	printf("Histogram[45] = %u\n",histogram[45]);
	printf("Histogram[56] = %u\n",histogram[56]);
	printf("Image[0] = %f\n",hostGrayImageData[0]);
	printf("Image[1] = %f\n",hostGrayImageData[1]);
	printf("Image[36] = %f\n",hostGrayImageData[36]);
	printf("Image[400] = %f\n",hostGrayImageData[400]);
	printf("Image[900] = %f\n",hostGrayImageData[900]);
	printf("Image[1405] = %f\n",hostGrayImageData[1405]);
	printf("Image[85000] = %f\n",hostGrayImageData[85000]);
	printf("First row of Gaussian filter = %f %f %f\n",filter[0][0], filter[0][1], filter[0][2]);
	printf("Second row of Gaussian filter = %f %f %f\n",filter[1][0], filter[1][1], filter[1][2]);
	printf("Third row of Gaussian filter = %f %f %f\n",filter[2][0], filter[2][1], filter[2][2]);
	printf("Blurred Image[0] = %f\n",BlurImageData[0]*255);
	printf("Blurred [25] = %f\n", BlurImageData[25]*255);
	printf("Blurred Image[290] = %f\n",BlurImageData[290]*255);
	printf("Gradient magnitude at [0] = %f\n",GradMagData[0]);
	printf("Gradient magnitude at [20] = %f\n",GradMagData[20]);
	printf("Gradient magnitude at [9000] = %f\n",GradMagData[9000]);
	printf("Gradient phase at [0] = %f\n",GradPhaseData[0]);
	printf("Gradient phase at [20] = %f\n",GradPhaseData[20]);
	printf("Gradient phase at [290] = %f\n",GradPhaseData[290]);
	printf("Otsu's Threshold = %f\n", thresh);
	printf("\n");

	// Export image
	char *oFile = wbArg_getOutputFile(args);
	wbExport(oFile, outputImage);


	//////////////
	// Clean Up //
	//////////////


	// Destory all cuda memory
	hipFree(deviceInputImageData);
	hipFree(deviceGrayImageData);
	hipFree(deviceBlurImageData);
	hipFree(deviceGradMagData);
	hipFree(deviceGradPhaseData);
	//hipFree(deviceEdgeImage);
	//hipFree(deviceWeakEdgeImage);

	// Destroy all host memory
	free(hostBlurImageData);
	free(hostGradMagData);
	free(hostGradPhaseData);
	//free(BlurImageData);
	free(GradMagData);
	free(GradPhaseData);
	free(NmsImageData);
	free(histogram);
	//free(EdgeImage);
	//free(WeakEdgeImage);

	// Destroy images
	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

	return 0;
}