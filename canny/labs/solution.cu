#include "hip/hip_runtime.h"
#include <stdio.h>
#include <wb.h>
#include "Otsus_Method.h"
#include "filters.h"
#include "non_max_supp.h"
#include "Edge_Connection.h"

#define FILTERSIZE 3

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

int main(int argc, char *argv[])
{

	//////////////////////////////
	// Parameter Initialization //
	//////////////////////////////


	// Image parameters for wbLib
	wbArg_t args;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	wbImage_t inputImage;
	wbImage_t outputImage;

	// Host side parameters
	float *hostInputImageData;
	float *hostGrayImageData; 
	float *hostBlurImageData;
	float *hostGradMagData;
	float *hostGradPhaseData;
	float *hostEdgeData;
	float *hostWeakEdgeData;

	// Device side parameters
	float *deviceInputImageData;
	float *deviceGrayImageData;
	float *deviceBlurImageData;
	float *deviceGradMagData;
	float *deviceGradPhaseData;
	float *deviceEdgeData;
	float *deviceWeakEdgeData;
	
	// Filtering parameters
	float *BlurImageData;
	float *GradMagData;
	float *GradPhaseData;
	float *NmsImageData;
	float *EdgeData;
	float *WeakEdgeData;

	// Otsu's Method parameters
	unsigned int *histogram;


	////////////////////
	// Image Handling //
	////////////////////


	// Parse the input arguments
	args = wbArg_read(argc, argv);

	// Read input file
	inputImageFile = wbArg_getInputFile(args, 0);

	// Import input image 
	inputImage = wbImport(inputImageFile);

	// Scrape info from input image
	imageWidth  = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage);
	
	// Define output image data
	hostInputImageData = wbImage_getData(inputImage);

	// Initialize memory for the output image
	// Note - input image is 3 channels. Other phases only have 1 channel
	// float *outData = (float *)calloc(imageHeight*imageWidth*imageChannels,sizeof(float));
	float *outData = (float *)calloc(imageHeight*imageWidth,sizeof(float));
	outputImage = wbImage_new(imageWidth, imageHeight, 1, outData);


	////////////////////////////////
	// Host Memory Initialization //
	////////////////////////////////


	// Allocate memory on host
	hostGrayImageData     = (float *)malloc(imageHeight*imageWidth*sizeof(float));
	hostBlurImageData     = (float *)malloc(imageHeight*imageWidth*sizeof(float));
	hostGradMagData 	  = (float *)malloc(imageHeight*imageWidth*sizeof(float));
	hostGradPhaseData 	  = (float *)malloc(imageHeight*imageWidth*sizeof(float));
	hostEdgeData 		  = (float *)malloc(imageHeight*imageWidth*sizeof(float));
	hostWeakEdgeData 	  = (float *)malloc(imageHeight*imageWidth*sizeof(float));

	// Allocate memory for serial filtering and initialize to 0
	BlurImageData       = (float *)calloc(imageHeight*imageWidth, sizeof(float));
	GradMagData    		= (float *)calloc(imageHeight*imageWidth, sizeof(float));
	GradPhaseData 		= (float *)calloc(imageHeight*imageWidth, sizeof(float));
	NmsImageData      	= (float *)calloc(imageHeight*imageWidth, sizeof(float));
	EdgeData	 		= (float *)calloc(imageHeight*imageWidth, sizeof(float));
	WeakEdgeData   		= (float *)calloc(imageHeight*imageWidth, sizeof(float));

	// Allocate memory on host and initialize to 0
	histogram = (unsigned int *)calloc(256, sizeof(unsigned int));
  

	/////////////////////////
	// Image Preprocessing //
	/////////////////////////


	// Create filter skeleton
	double filter[FILTERSIZE][FILTERSIZE];

	// Fill the gaussian filter
	populate_blur_filter(filter);

	// ?????
	int filterSize = (int)FILTERSIZE;


	//////////////////////////////////
	// Device Memory Initialization //
	//////////////////////////////////


	// Start total program timer
	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	// Start memory allocation timer
	wbTime_start(GPU, "Doing GPU memory allocation");

	// Allocate memory on device
	hipMalloc((void **)&deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **)&deviceGrayImageData, imageWidth*imageHeight*sizeof(float));
	hipMalloc((void **)&deviceBlurImageData, imageWidth*imageHeight*sizeof(float));
	hipMalloc((void **)&deviceGradMagData, imageWidth*imageHeight*sizeof(float));
	hipMalloc((void **)&deviceGradPhaseData, imageWidth*imageHeight*sizeof(float));
	hipMalloc((void **)&deviceEdgeData, imageWidth*imageHeight*sizeof(float));
	hipMalloc((void **)&deviceWeakEdgeData, imageWidth*imageHeight*sizeof(float));

	// Stop memory allocation timer
	wbTime_stop(GPU, "Doing GPU memory allocation");

	// Start memory copy timer
	wbTime_start(Copy, "Copying data to the GPU");

	// Copy input image from host to device
	hipMemcpy(deviceInputImageData, hostInputImageData, imageChannels*imageWidth*imageHeight*sizeof(float), hipMemcpyHostToDevice);


	///////////////////
	// GPU Execution //
	///////////////////


	// Start computation timer
	wbTime_start(Compute, "Doing the computation on the GPU");

	// Number of threads/block is 16
	int blocksize = 16;

	// Initialize x and y block dimension to blocksize
	dim3 BlockDim(blocksize,blocksize);

	// Set x and y grid dimension 
	dim3 GridDim(((imageWidth+BlockDim.x-1)/BlockDim.x), ((imageHeight+BlockDim.y-1)/BlockDim.y));  

	// Call RGB to grayscale conversion kernel
	ColorToGrayscale<<<GridDim, BlockDim>>>(deviceInputImageData, deviceGrayImageData, imageWidth, imageHeight);

	// Call image burring kernel
	//Conv2D<<<GridDim, BlockDim>>>(deviceGrayImageData, deviceBlurImageData, filter, imageWidth, imageHeight, filterSize);

	// Call sobel filtering kernel
	//GradientSobel<<<GridDim, BlockDim>>>(deviceBlurImageData, deviceSobelImageData, deviceSobelImageData, imageHeight, imageWidth); 

	// Stop computation timer
	wbTime_stop(Compute, "Doing the computation on the GPU");


	////////////////////
	// Device Results //
	////////////////////


	// Start device memory copy timer
	wbTime_start(Copy, "Copying data from the GPU");

	// Copy data from device back to host
	hipMemcpy(hostGrayImageData, deviceGrayImageData, imageWidth*imageHeight*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(hostBlurImageData, deviceBlurImageData, imageWidth*imageHeight*sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(hostGrayImageData, deviceGrayImageData, imageWidth*imageHeight*sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(hostSobelImageData, deviceSobelImageData, imageWidth*imageHeight*sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(hostGradientImageData, deviceGradientImageData, imageWidth*imageHeight*sizeof(float), hipMemcpyHostToDevice); 

	// Stop memory timer
	wbTime_stop(Copy, "Copying data from the GPU");

	// Stop total program timer
	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	////////////////////
	// Host Execution //
	////////////////////

	// Blur image using Gaussian Kernel
	Conv2DSerial(hostGrayImageData, BlurImageData, filter, imageWidth, imageHeight, filterSize);

	// Calculate gradient using Sobel Operators
	GradientSobelSerial(BlurImageData, GradMagData, GradPhaseData, imageHeight, imageWidth);

	// Suppress non-maximum pixels along gradient
	nms(GradMagData, NmsImageData, GradPhaseData, imageHeight, imageWidth);

	// Calculate histogram of blurred image
	Histogram_Sequential(NmsImageData, histogram, imageWidth, imageHeight);

	// Calculate threshold using Otsu's Method
	double thresh = Otsu_Sequential(histogram);

	threshold_detection_serial(NmsImageData, WeakEdgeData, EdgeData, thresh, imageWidth, imageHeight);

	edge_connection_serial(WeakEdgeData, EdgeData, imageWidth, imageHeight);

	

	////////////////////////
	// Logging and Output //
	////////////////////////


	// Copy image data for output image (choose 1 - can only log one at a time for now
	// For GPU execution
	//memcpy(outData, hostGrayImageData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, hostBlurImageData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, hostGradMagData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, hostGradPhaseData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, hostNmsImageData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, hostWeakEdgeData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, hostEdgeData, imageHeight*imageWidth*sizeof(float));

	// For Host execution
	//memcpy(outData, GrayImageData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, BlurImageData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, GradMagData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, GradPhaseData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, NmsImageData, imageHeight*imageWidth*sizeof(float));
	//memcpy(outData, WeakEdgeData, imageHeight*imageWidth*sizeof(float));
	memcpy(outData, EdgeData, imageHeight*imageWidth*sizeof(float));

	// Export image
	char *oFile = wbArg_getOutputFile(args);
	wbExport(oFile, outputImage);


	////////////////////
	// Debugging Info //
	////////////////////


	// Print info
	printf("\n");
	printf("Width = %u\n",imageWidth);
	printf("Height = %u\n",imageHeight);
	printf("InputImage[0] = %f\n",hostInputImageData[0]);
	printf("Histogram[0] = %u\n",histogram[0]);
	printf("Histogram[1] = %u\n",histogram[1]);
	printf("Histogram[20] = %u\n",histogram[20]);
	printf("Histogram[45] = %u\n",histogram[45]);
	printf("Histogram[56] = %u\n",histogram[56]);
	printf("Histogram[255] = %u\n",histogram[255]);
	printf("Image[0] = %f\n",hostGrayImageData[0]);
	printf("Image[1] = %f\n",hostGrayImageData[1]);
	printf("Image[36] = %f\n",hostGrayImageData[36]);
	printf("Image[400] = %f\n",hostGrayImageData[400]);
	printf("Image[900] = %f\n",hostGrayImageData[900]);
	printf("Image[1405] = %f\n",hostGrayImageData[1405]);
	printf("Image[85000] = %f\n",hostGrayImageData[85000]);
	printf("First row of Gaussian filter = %f %f %f\n",filter[0][0], filter[0][1], filter[0][2]);
	printf("Second row of Gaussian filter = %f %f %f\n",filter[1][0], filter[1][1], filter[1][2]);
	printf("Third row of Gaussian filter = %f %f %f\n",filter[2][0], filter[2][1], filter[2][2]);
	printf("Blurred Image[0] = %f\n",BlurImageData[0]*255);
	printf("Blurred [25] = %f\n", BlurImageData[25]*255);
	printf("Blurred Image[290] = %f\n",BlurImageData[290]*255);
	printf("Gradient magnitude at [0] = %f\n",GradMagData[0]);
	printf("Gradient magnitude at [20] = %f\n",GradMagData[20]);
	printf("Gradient magnitude at [9000] = %f\n",GradMagData[9000]);
	printf("Gradient phase at [0] = %f\n",GradPhaseData[0]);
	printf("Gradient phase at [20] = %f\n",GradPhaseData[20]);
	printf("Gradient phase at [290] = %f\n",GradPhaseData[290]);
	printf("NMS at [0] = %f\n",NmsImageData[0]);
	printf("NMS at [20] = %f\n",NmsImageData[20]);
	printf("NMS at [130] = %f\n",NmsImageData[130]);
	printf("NMS at [131] = %f\n",NmsImageData[131]);
	printf("Otsu's Threshold = %f\n", thresh);
	printf("\n");


	//////////////
	// Clean Up //
	//////////////


	// Destory all cuda memory
	hipFree(deviceInputImageData);
	hipFree(deviceGrayImageData);
	hipFree(deviceBlurImageData);
	hipFree(deviceGradMagData);
	hipFree(deviceGradPhaseData);
	hipFree(deviceEdgeData);
	hipFree(deviceWeakEdgeData);

	// Destroy host memory
	free(hostBlurImageData);
	free(hostGradMagData);
	free(hostGradPhaseData);
	free(hostEdgeData);
	free(hostWeakEdgeData);

	// Destroy CPU memory
	free(BlurImageData);
	free(GradMagData);
	free(GradPhaseData);
	free(NmsImageData);
	free(histogram);
	free(EdgeData);
	free(WeakEdgeData);

	// Destroy images
	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

	return 0;
}
