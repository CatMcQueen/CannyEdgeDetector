#include "hip/hip_runtime.h"
#include <wb.h>
#include "filters.cu"
#include "Otsus_Method.h"

#define FILTERSIZE 3

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

void populate_blur_filter(double outFilter[FILTERSIZE][FILTERSIZE])
{
    double scaleVal = 1;
    double stDev = (double)FILTERSIZE/3;

    for (int i = 0; i < FILTERSIZE; ++i) {
        for (int j = 0; j < FILTERSIZE; ++j) {
            double xComp = pow((i - FILTERSIZE/2), 2);
            double yComp = pow((j - FILTERSIZE/2), 2);

            double stDevSq = pow(stDev, 2);
            double pi = M_PI;

            //calculate the value at each index of the Kernel
            double filterVal = exp(-(((xComp) + (yComp)) / (2 * stDevSq)));
            filterVal = (1 / (sqrt(2 * pi)*stDev)) * filterVal;

            //populate Kernel
            outFilter[i][j] =filterVal;

            if (i==0 && j==0) 
            {
                scaleVal = outFilter[0][0];
            }

            //normalize Kernel
            outFilter[i][j] = outFilter[i][j] / scaleVal;
        }
    }
}

int main(int argc, char *argv[])
{

	//////////////////////////////
	// Parameter Initialization //
	//////////////////////////////


	// Image parameters for wbLib
	wbArg_t args;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	wbImage_t inputImage;
	wbImage_t outputImage;

	// Host side parameters
	float *hostInputImageData;
	float *hostGrayImageData;
	float *hostBlurImageData;
	float *hostGradientImageData;
	float *hostSobelImageData;

	// Device side parameters
	float *deviceInputImageData;
	float *deviceGrayImageData;
	float *deviceBlurImageData;
	float *deviceGradientImageData;
	float *deviceSobelImageData;
	
	// Otsu's Method parameters
	unsigned int *histogram;


	////////////////////
	// Image Handling //
	////////////////////


	// Parse the input arguments
	args = wbArg_read(argc, argv);

	// Read input file
	inputImageFile = wbArg_getInputFile(args, 0);

	// Import input image 
	inputImage = wbImport(inputImageFile);

	// Scrape info from input image
	imageWidth  = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage);
	
	// Define new output image
	outputImage = wbImage_new(imageWidth, imageHeight, 1);

	// Define output image data
	hostInputImageData = wbImage_getData(inputImage);

	// CHANGE THIS TO CHANGE OUTPUT IMAGE
	hostGrayImageData = wbImage_getData(outputImage);


	////////////////////////////////
	// Host Memory Initialization //
	////////////////////////////////


	// Allocate memory on host
	hostBlurImageData     = (float *)malloc(imageHeight*imageWidth*sizeof(float));
	hostSobelImageData    = (float *)malloc(imageHeight*imageWidth*sizeof(float));
	hostGradientImageData = (float *)malloc(imageHeight*imageWidth*sizeof(float));

	// Allocate memory on host and set to 0
	histogram = (unsigned int *)calloc(256, sizeof(unsigned int));


	/////////////////////////
	// Image Preprocessing //
	/////////////////////////


	// Create filter skeleton
	double filter[FILTERSIZE][FILTERSIZE];

	// Fill the gaussian filter
	populate_blur_filter(filter);

	// ?????
	int filterSize = (int)FILTERSIZE;


	//////////////////////////////////
	// Device Memory Initialization //
	//////////////////////////////////


	// Start total program timer
	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	// Start memory allocation timer
	wbTime_start(GPU, "Doing GPU memory allocation");

	// Allocate memory on device
	hipMalloc((void **)&deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **)&deviceGrayImageData, imageWidth*imageHeight*sizeof(float));
	hipMalloc((void **)&deviceBlurImageData, imageWidth*imageHeight*sizeof(int));
	hipMalloc((void **)&deviceSobelImageData, imageWidth*imageHeight*sizeof(int));
	hipMalloc((void **)&deviceGradientImageData, imageWidth*imageHeight*sizeof(int));

	// Stop memory allocation timer
	wbTime_stop(GPU, "Doing GPU memory allocation");

	// Start memory copy timer
	wbTime_start(Copy, "Copying data to the GPU");

	// Copy input image from host to device
	hipMemcpy(deviceInputImageData, hostInputImageData, imageChannels*imageWidth*imageHeight*sizeof(float), hipMemcpyHostToDevice);


	///////////////////
	// GPU Execution //
	///////////////////


	// Start computation timer
	wbTime_start(Compute, "Doing the computation on the GPU");

	// Number of threads/block is 16
	int blocksize = 16;

	// Initialize x and y block dimension to blocksize
	dim3 BlockDim(blocksize,blocksize);

	// Set x and y grid dimension 
	dim3 GridDim(((imageWidth+BlockDim.x-1)/BlockDim.x), ((imageHeight+BlockDim.y-1)/BlockDim.y));  

	// Call RGB to grayscale conversion kernel
	ColorToGrayscale<<<GridDim, BlockDim>>>(deviceInputImageData, deviceGrayImageData, imageWidth, imageHeight);

	// Call image burring kernel
	//Conv2D<<<GridDim, BlockDim>>>(deviceGrayImageData, deviceBlurImageData, filter, imageWidth, imageHeight, filterSize);

	// Call sobel filtering kernel
	//GradientSobel<<<GridDim, BlockDim>>>(deviceBlurImageData, deviceSobelImageData, deviceSobelImageData, imageHeight, imageWidth); 

	// Stop computation timer
	wbTime_stop(Compute, "Doing the computation on the GPU");


	////////////////////
	// Device Results //
	////////////////////


	// Start device memory copy timer
	wbTime_start(Copy, "Copying data from the GPU");

	// Copy data from device back to host
	hipMemcpy(hostGrayImageData, deviceGrayImageData, imageWidth*imageHeight*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(hostBlurImageData, deviceBlurImageData, imageWidth*imageHeight*sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(hostGrayImageData, deviceGrayImageData, imageWidth*imageHeight*sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(hostSobelImageData, deviceSobelImageData, imageWidth*imageHeight*sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(hostGradientImageData, deviceGradientImageData, imageWidth*imageHeight*sizeof(float), hipMemcpyHostToDevice); 

	// Stop memory timer
	wbTime_stop(Copy, "Copying data from the GPU");

	// Stop total program timer
	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	
	////////////////////
	// Host Execution //
	////////////////////


	// Calculate histogram of blurred image
	Histogram_Sequential(hostGrayImageData, histogram, imageWidth, imageHeight);

	// Calculate threshold using Otsu's Method
	double thresh = Otsu_Sequential(histogram);


	////////////////////
	// Debugging Info //
	////////////////////


	// Print info
	printf("\n");
	printf("Width = %u\n",imageWidth);
	printf("Height = %u\n",imageHeight);
	printf("InputImage[0] = %f\n",hostInputImageData[0]);
	printf("Histogram[0] = %u\n",histogram[0]);
	printf("Histogram[1] = %u\n",histogram[1]);
	printf("Histogram[20] = %u\n",histogram[20]);
	printf("Histogram[45] = %u\n",histogram[45]);
	printf("Histogram[56] = %u\n",histogram[56]);
	printf("Image[0] = %f\n",hostGrayImageData[0]);
	printf("Image[1] = %f\n",hostGrayImageData[1]);
	printf("Image[36] = %f\n",hostGrayImageData[36]);
	printf("Image[400] = %f\n",hostGrayImageData[400]);
	printf("Image[900] = %f\n",hostGrayImageData[900]);
	printf("Image[1405] = %f\n",hostGrayImageData[1405]);
	printf("Image[85000] = %f\n",hostGrayImageData[85000]);
	printf("Otsu's Threshold = %f\n", thresh);
	printf("\n");

	// Export image
	char *oFile = wbArg_getOutputFile(args);
	wbExport(oFile, outputImage);


	//////////////
	// Clean Up //
	//////////////


	// Destory all cuda memory
	hipFree(deviceInputImageData);
	hipFree(deviceGrayImageData);
	hipFree(deviceBlurImageData);
	hipFree(deviceSobelImageData);
	hipFree(deviceGradientImageData);

	// Destroy all host memory
	free(hostBlurImageData);
	free(hostSobelImageData);
	free(hostGradientImageData);
	free(histogram);

	// Destroy images
	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

	return 0;
}
