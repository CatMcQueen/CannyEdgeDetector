#include "hip/hip_runtime.h"
#include "Otsus_Method.h"
#include <cmath>

#define NUM_BINS 256

void Histogram_Sequential(float *image, unsigned int *hist, int width, int height)
{
	int pos = 0;

	// Loop through every pixel
	for (int row = 0; row < height; row++)
	{
		for (int col = 0; col < width; col++)
		{
			if (image[row*width + col] > 1)
			{
				pos = 255;
			}
			else
			{
				pos = int(image[row*width + col] * 255);
			}

			// Update histogram
			hist[pos]++;
		}

	}

}

double Otsu_Sequential(unsigned int* histogram, int width, int height)
{

	float bin_mids[256];
	float histogram_bin_mids[256];
	float weight1[256];
	float weight2[256];
	float cumsum_mean1[256];
	float cumsum_mean2[256];
	float mean1[256];
	float mean2[256];
	float inter_class_variance[255];
	float max_variance = 0;

	int thresh = 0;

	float bin_length = 255.0f/256.0f;
	float half_bin_length = 255.0f/512.0f;

	// Calculate bin mids
	for(int i = 0; i < 256; i++)
	{
		bin_mids[i] = half_bin_length + bin_length * i;
		histogram_bin_mids[i] = histogram[i] * (half_bin_length + bin_length * i);
	}

	weight1[0] = histogram[0];
	weight2[0] = width * height;

	// Calculate class probabilities
	for(int i = 1; i < 256; i++)
	{
		weight1[i] = histogram[i] + weight1[i-1];
		weight2[i] = weight2[i-1] - histogram[i-1];
	}

	cumsum_mean1[0] = histogram_bin_mids[0];
	cumsum_mean2[0] = histogram_bin_mids[255];

	// Calculate class means
	for(int i = 1; i < 256; i++)
	{
		cumsum_mean1[i] = cumsum_mean1[i-1] + histogram_bin_mids[i];
		cumsum_mean2[i] = cumsum_mean2[i-1] + histogram_bin_mids[256 - i - 1];
		mean1[i] = cumsum_mean1[i] / weight1[i];
		mean2[256 - i - 1] = cumsum_mean2[i] / weight2[256 - i - 1];
	}

	// Calculate Inter_class_variance
	for(int i = 0; i < 255; i++)
	{
		inter_class_variance[i] = (weight1[i] * weight2[i] * (mean1[i] - mean2[i+1])) * (mean1[i] - mean2[i+1]);	
	}

	// Maximize interclass variance
	for(int i = 0;i < 255; i++){
		if(max_variance < inter_class_variance[i])
		{
			max_variance = inter_class_variance[i];
			thresh = i;
		}
	}

	// Return normalized threshold
	return bin_mids[thresh];

}

__global__ void NaiveHistogram(float* image, unsigned int* histogram, int width, int height)
{
	// insert your code here
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int stride = blockDim.x * gridDim.x;

	while(tid < width * height)
	{
		int position = int(image[tid]*255);

		if (position >= 0 && position < 256)
		{
			atomicAdd(&(histogram[position]),1);
		}

		tid += stride;

	}
}

__global__ void OptimizedHistogram(float* image, unsigned int* histogram, int width, int height)
{
	// insert your code here
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int stride = blockDim.x * gridDim.x;

	while(tid < width * height)
	{
		int position = int(image[tid]*255);

		if (position >= 0 && position < 256)
		{
			atomicAdd(&(histogram[position]),1);
		}

		tid += stride;

	}
}

__global__ void NaiveOtsu(unsigned int *histogram, float* thresh, int width, int height)
{
	__shared__ float weight1[256];
	__shared__ float weight2[256];

	__shared__ float bin_mids[256];
	__shared__ float histogram_bin_mids[256];

	__shared__ float mean1[256];
	__shared__ float mean2[257];

	__shared__ float inter_class_variance[256];
	__shared__ int key[256];

	float bin_length = 0.99609375;
	float half_bin_length = 0.498046875;

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < 256)
	{
		bin_mids[tid] = half_bin_length + bin_length * tid;
		histogram_bin_mids[tid] = histogram[tid] * (half_bin_length + bin_length * tid);

		__syncthreads();

		float w1 = histogram[0];
		float w2 = width * height;

		float cs_mean1 = histogram_bin_mids[0];
		float cs_mean2 = histogram_bin_mids[255];

		// Calculate class probabilities and means
		for(int i = 1; i < tid + 1; i++)
		{
			w1 += histogram[i];
			w2 -= histogram[i-1];
			cs_mean1 += histogram_bin_mids[i];
			cs_mean2 += histogram_bin_mids[256-i-1];
		}

		weight1[tid] = w1;
		weight2[tid] = w2;

		__syncthreads();

		mean1[tid] = cs_mean1 / weight1[tid];
		mean2[256 - tid - 1] = cs_mean2 / weight2[256 - tid - 1];

		if (tid == 0)
		{
			mean1[0] = 0;
		}

		if (tid == 255)
		{
			mean2[255] = 0;
		}
	
		key[tid] = tid;

		__syncthreads();

		inter_class_variance[tid] = (weight1[tid] * weight2[tid] * (mean1[tid] - mean2[tid+1])) * (mean1[tid] - mean2[tid+1]);

		for (int stride = 1; stride < 256; stride *= 2)
		{
			if(tid % (2*stride) == 0)
			{
				if(inter_class_variance[tid] < inter_class_variance[tid+stride])
				{
					inter_class_variance[tid] = inter_class_variance[tid+stride];
					key[tid] = key[tid+stride];
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if(tid == 0)
	{
		thresh[0] = bin_mids[key[0]];
	}

}


__global__ void OptimizedOtsu(unsigned int *histogram, float* thresh, int width, int height)
{
	__shared__ float weight1[256];
	__shared__ float weight2[256];

	__shared__ float bin_mids[256];
	__shared__ float histogram_bin_mids[256];

	__shared__ float mean1[256];
	__shared__ float mean2[257];

	__shared__ float inter_class_variance[256];
	__shared__ int key[256];

	float bin_length = 0.99609375;
	float half_bin_length = 0.498046875;

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < 256)
	{
		bin_mids[tid] = half_bin_length + bin_length * tid;
		histogram_bin_mids[tid] = histogram[tid] * (half_bin_length + bin_length * tid);

		__syncthreads();

		float w1 = histogram[0];
		float w2 = width * height;

		float cs_mean1 = histogram_bin_mids[0];
		float cs_mean2 = histogram_bin_mids[255];

		// Calculate class probabilities and means
		for(int i = 1; i < tid + 1; i++)
		{
			w1 += histogram[i];
			w2 -= histogram[i-1];
			cs_mean1 += histogram_bin_mids[i];
			cs_mean2 += histogram_bin_mids[256-i-1];
		}

		weight1[tid] = w1;
		weight2[tid] = w2;

		__syncthreads();

		mean1[tid] = cs_mean1 / weight1[tid];
		mean2[256 - tid - 1] = cs_mean2 / weight2[256 - tid - 1];

		if (tid == 0)
		{
			mean1[0] = 0;
		}

		if (tid == 255)
		{
			mean2[255] = 0;
		}
	
		key[tid] = tid;

		__syncthreads();

		inter_class_variance[tid] = (weight1[tid] * weight2[tid] * (mean1[tid] - mean2[tid+1])) * (mean1[tid] - mean2[tid+1]);

		for (int stride = 1; stride < 256; stride *= 2)
		{
			if(tid % (2*stride) == 0)
			{
				if(inter_class_variance[tid] < inter_class_variance[tid+stride])
				{
					inter_class_variance[tid] = inter_class_variance[tid+stride];
					key[tid] = key[tid+stride];
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if(tid == 0)
	{
		thresh[0] = bin_mids[key[0]];
	}

}