#include "hip/hip_runtime.h"
#include "non_max_supp.h"
//#include <stdio.h>

__host__ __device__ float maxSupp(float center, float p1, float p2, float p3 = -1.0, float p4 = -1.0)
{
    if (center >= p1 && center >= p2 && center >= p3 && center >= p4)
    {
        return center;
    }
    else
    {
        return 0.0;
    }
}

__host__ __device__ float getPoint(float *img, int cIdx, int rIdx, int height, int width)
{
    if (!img || rIdx < 0 || rIdx >= height || cIdx < 0 || cIdx >= width)
    {
        return 0.0;
    }
    return *(img + cIdx + rIdx * width);
}

void nms(float *inImg, float *nmsImg, float *gradImg, int height, int width)
{
    // FILE *quantFile = fopen("quantNms.txt", "w");

    for (int j = 0; j < height; ++j)
    {
        for (int i = 0; i < width; ++i)
        {
            float angle = *(gradImg + j * width + i);
            float p1 = -1.0; //, p3 = -1.0;
            float p2 = -1.0; //, p4 = -1.0;
            unsigned int fAngle = 0;
            if (angle > 180)
            {
                angle = angle - 180;
            }

            // if ((angle > 0 && angle <= 22.5) || (angle > 157.5 && angle <= 180))
            //   fAngle = 0;
            // else if (angle > 22.5 && angle <= 67.5)
            //   fAngle = 45;
            // else if (angle > 67.5 && angle <= 112.5)
            //   fAngle = 90;
            // else if (angle > 112.5 && angle <= 157.5)
            //   fAngle = 135;

            if ((angle > -22.5 && angle <= 22.5) || (angle > 157.5) || (angle < -157.5))
                fAngle = 0;
            else if ((angle > 112.5 && angle <= 157.5) || (angle < -22.5 && angle >= -67.5))
                fAngle = 135;
            else if ((angle > 67.5 && angle <= 112.5) || (angle < -67.5 && angle >= -112.5))
                fAngle = 90;
            else if ((angle > 22.5 && angle <= 67.5) || (angle < -112.5 && angle >= -157.5))
                fAngle = 45;

            // fprintf(quantFile, "%d,", fAngle);
            switch (fAngle)
            {
            case 0:
                p1 = getPoint(inImg, i, j + 1, height, width);
                p2 = getPoint(inImg, i, j - 1, height, width);
                // p3 = getPoint(inImg, i, j+2, height, width);
                // p4 = getPoint(inImg, i, j-2, height, width);
                break;
            case 45:
                p1 = getPoint(inImg, i - 1, j - 1, height, width);
                p2 = getPoint(inImg, i + 1, j + 1, height, width);
                // p3 = getPoint(inImg, i-2, j-2, height, width);
                // p4 = getPoint(inImg, i+2, j+2, height, width);
                break;
            case 90:
                p1 = getPoint(inImg, i + 1, j, height, width);
                p2 = getPoint(inImg, i - 1, j, height, width);
                // p3 = getPoint(inImg, i+2, j, height, width);
                // p4 = getPoint(inImg, i-2, j, height, width);
                break;
            case 135:
                p1 = getPoint(inImg, i + 1, j - 1, height, width);
                p2 = getPoint(inImg, i - 1, j + 1, height, width);
                // p3 = getPoint(inImg, i+2, j-2, height, width);
                // p4 = getPoint(inImg, i-2, j+2, height, width);
                break;
            default:
                break;
            }

            float center = getPoint(inImg, i, j, height, width);
            //*(nmsImg + i + j*width) = maxSupp(center, p1, p2, p3, p4);
            *(nmsImg + i + j * width) = maxSupp(center, p1, p2);
        }
        // fprintf(quantFile, "\n");
    }
    // fclose(quantFile);
}

__global__ void nms_global(float *inImg, float *nmsImg, float *gradImg, int height, int width)
{
    size_t col = blockDim.x * blockIdx.x + threadIdx.x;
    size_t row = blockDim.y * blockIdx.y + threadIdx.y;

    float p1 = -1.0; //, p3 = -1.0;
    float p2 = -1.0; //, p4 = -1.0;
    unsigned int fAngle = 0;

    // if (col >= 0 && col < width && row >= 0 && row < height)
    if (col < width && row < height) // Since size_t is unsigned, it can't fall below 0
    {
        float angle = *(gradImg + row * width + col);

        if ((angle > -22.5 && angle <= 22.5) || (angle > 157.5) || (angle < -157.5))
            fAngle = 0;
        else if ((angle > 112.5 && angle <= 157.5) || (angle < -22.5 && angle >= -67.5))
            fAngle = 135;
        else if ((angle > 67.5 && angle <= 112.5) || (angle < -67.5 && angle >= -112.5))
            fAngle = 90;
        else if ((angle > 22.5 && angle <= 67.5) || (angle < -112.5 && angle >= -157.5))
            fAngle = 45;

        switch (fAngle)
        {
        case 0:
            p1 = getPoint(inImg, col, row + 1, height, width);
            p2 = getPoint(inImg, col, row - 1, height, width);
            // p3 = getPoint(inImg, col, row+2, height, width);
            // p4 = getPoint(inImg, col, row-2, height, width);
            break;
        case 45:
            p1 = getPoint(inImg, col - 1, row - 1, height, width);
            p2 = getPoint(inImg, col + 1, row + 1, height, width);
            // p3 = getPoint(inImg, col-2, row-2, height, width);
            // p4 = getPoint(inImg, col+2, row+2, height, width);
            break;
        case 90:
            p1 = getPoint(inImg, col + 1, row, height, width);
            p2 = getPoint(inImg, col - 1, row, height, width);
            // p3 = getPoint(inImg, col+2, row, height, width);
            // p4 = getPoint(inImg, col-2, row, height, width);
            break;
        case 135:
            p1 = getPoint(inImg, col + 1, row - 1, height, width);
            p2 = getPoint(inImg, col - 1, row + 1, height, width);
            // p3 = getPoint(inImg, col+2, row-2, height, width);
            // p4 = getPoint(inImg, col-2, row+2, height, width);
            break;
        default:
            break;
        }

        float center = getPoint(inImg, col, row, height, width);
        //*(nmsImg + i + j*width) = maxSupp(center, p1, p2, p3, p4);
        *(nmsImg + col + row * width) = maxSupp(center, p1, p2);
    }
}

__global__ void nms_opt(float *inImg, float *nmsImg, float *gradImg, int height, int width)
{
    // pixel 299, 299
    // blockIdx = 300.0 / 16 = 18.75
    // 18*16 = 288 --> thread 11
    size_t col = blockDim.x * blockIdx.x + threadIdx.x;
    size_t row = blockDim.y * blockIdx.y + threadIdx.y;

    float p1 = -1.0; //, p3 = -1.0;
    float p2 = -1.0; //, p4 = -1.0;

    const size_t TILE_SIZE = 16;
    const size_t P_IMG_SIZE = TILE_SIZE + 2; // Handle overrun on edges
    __shared__ float pImage[TILE_SIZE + 2][TILE_SIZE + 2];
    __shared__ float pAngle[TILE_SIZE][TILE_SIZE];

    if (col < width + 2 && row < height + 2) // Since size_t is unsigned, it can't fall below 0
    {
        // for(size_t i = 0; i < P_IMG_SIZE; i += TILE_SIZE)
        for (size_t i = 0; threadIdx.x + i < P_IMG_SIZE; i += TILE_SIZE)
        {
            for (size_t j = 0; threadIdx.y + j < P_IMG_SIZE; j += TILE_SIZE)
            {
                pImage[threadIdx.x + i][threadIdx.y + j] = getPoint(inImg, col + i - 1, row + j - 1, height, width);
            }
        }
        pAngle[threadIdx.x][threadIdx.y] = getPoint(gradImg, col, row, height, width); // gradImg[row*width + col];
    }
    __syncthreads();

    if (col < width && row < height) // Since size_t is unsigned, it can't fall below 0
    {
        float angle = pAngle[threadIdx.x][threadIdx.y]; //*(gradImg + row*width + col);
        size_t i = threadIdx.x + 1;
        size_t j = threadIdx.y + 1;

        if ((angle > -22.5 && angle <= 22.5) || (angle > 157.5) || (angle < -157.5))
        {
            p1 = pImage[i][j + 1];
            p2 = pImage[i][j - 1];
            // p3 = getPoint(inImg, i, j+2, 16, 16);
            // p4 = getPoint(inImg, i, j-2, 16, 16);
        }
        else if ((angle > 112.5 && angle <= 157.5) || (angle < -22.5 && angle >= -67.5))
        {
            p1 = pImage[i + 1][j - 1];
            p2 = pImage[i - 1][j + 1];
            // p3 = getPoint(inImg, i+2, j-2, 16, 16);
            // p4 = getPoint(inImg, i-2, j+2, 16, 16);
        }
        else if ((angle > 67.5 && angle <= 112.5) || (angle < -67.5 && angle >= -112.5))
        {
            p1 = pImage[i + 1][j];
            p2 = pImage[i - 1][j];
            // p3 = getPoint(inImg, i+2, j, 16, 16);
            // p4 = getPoint(inImg, i-2, j, 16, 16);
        }
        else if ((angle > 22.5 && angle <= 67.5) || (angle < -112.5 && angle >= -157.5))
        {
            p1 = pImage[i - 1][j - 1];
            p2 = pImage[i + 1][j + 1];
            // p3 = getPoint(inImg, i-2, j-2, 16, 16);
            // p4 = getPoint(inImg, i+2, j+2, 16, 16);
        }

        float center = pImage[i][j];
        ////*(nmsImg + i + j*width) = maxSupp(center, p1, p2, p3, p4);
        //*(nmsImg + col + row*width) = maxSupp(center, p1, p2);
        // for(size_t i = threadIdx.x + 1; i < TILE_SIZE + 1; i += TILE_SIZE)
        // for(size_t i = threadIdx.x; i < TILE_SIZE; i += TILE_SIZE)
        //{
        //  //for(size_t j = threadIdx.y + 1; j < TILE_SIZE + 1; j+= TILE_SIZE)
        //  for(size_t j = threadIdx.y; j < TILE_SIZE; j+= TILE_SIZE)
        //  {
        //    //nmsImg[col + i - 1 + (row + j - 1)*width] = pImage[i][j];
        //    nmsImg[(col + i) + (row + j)*width] = pImage[i][j];
        //  }
        //}
        // nmsImg[col + row*width] = pImage[threadIdx.x + 1][threadIdx.y + 1];
        nmsImg[col + row * width] = maxSupp(center, p1, p2);
    }
}
