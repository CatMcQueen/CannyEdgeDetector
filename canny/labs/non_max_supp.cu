#include "hip/hip_runtime.h"
#include "non_max_supp.h"
//#include <stdio.h>

__host__ __device__
float maxSupp(float center, float p1, float p2, float p3=-1.0, float p4=-1.0)
{
  if (center >= p1 && center >= p2 && center >= p3 && center >= p4)
  {
    return center;
  }
  else
  {
    return 0.0;
  }
}

__host__ __device__
float getPoint(float *img, int cIdx, int rIdx, int height, int width)
{
  if (!img || rIdx < 0 || rIdx >= width || cIdx < 0 || cIdx >= height)
  {
    return 0.0;
  }
  return *(img + cIdx + rIdx*width);
}

void nms(float *inImg, float *nmsImg, float *gradImg, int height, int width)
{
  //FILE *quantFile = fopen("quantNms.txt", "w");

  for(int j = 0; j < height; ++j)
  {
    for(int i = 0; i < width; ++i)
    {
      float angle = *(gradImg + j*width + i);
      float p1 = -1.0, p2 = -1.0;
      float p3 = -1.0, p4 = -1.0;
      unsigned int fAngle = 0;
      if (angle > 180)
      {
        angle = angle - 180;
      }

      //if ((angle > 0 && angle <= 22.5) || (angle > 157.5 && angle <= 180))
      //  fAngle = 0;
      //else if (angle > 22.5 && angle <= 67.5)
      //  fAngle = 45;
      //else if (angle > 67.5 && angle <= 112.5)
      //  fAngle = 90;
      //else if (angle > 112.5 && angle <= 157.5)
      //  fAngle = 135;

      if ((angle > -22.5 && angle <= 22.5) || (angle > 157.5) || (angle < -157.5))
        fAngle = 0;
      else if ((angle > 112.5 && angle <= 157.5) || (angle < -22.5 && angle >= -67.5))
        fAngle = 135;
      else if ((angle > 67.5 && angle <= 112.5) || (angle < -67.5 && angle >= -112.5))
        fAngle = 90;
      else if ((angle > 22.5 && angle <= 67.5) || (angle < -112.5 && angle >= -157.5))
        fAngle = 45;
      
      //fprintf(quantFile, "%d,", fAngle);
      switch( fAngle ) 
      {
        case 0:
          p1 = getPoint(inImg, i, j+1, height, width);
          p2 = getPoint(inImg, i, j-1, height, width);
          p3 = getPoint(inImg, i, j+2, height, width);
          p4 = getPoint(inImg, i, j-2, height, width);
          break;
        case 45:
          p1 = getPoint(inImg, i-1, j-1, height, width);
          p2 = getPoint(inImg, i+1, j+1, height, width);
          p3 = getPoint(inImg, i-2, j-2, height, width);
          p4 = getPoint(inImg, i+2, j+2, height, width);
          break;
        case 90:
          p1 = getPoint(inImg, i+1, j, height, width);
          p2 = getPoint(inImg, i-1, j, height, width);
          p3 = getPoint(inImg, i+2, j, height, width);
          p4 = getPoint(inImg, i-2, j, height, width);
          break;
        case 135:
          p1 = getPoint(inImg, i+1, j-1, height, width);
          p2 = getPoint(inImg, i-1, j+1, height, width);
          p3 = getPoint(inImg, i+2, j-2, height, width);
          p4 = getPoint(inImg, i-2, j+2, height, width);
          break;
        default:
          break;
      }

      float center = getPoint(inImg, i, j, height, width);
      //*(nmsImg + i + j*width) = maxSupp(center, p1, p2, p3, p4);
      *(nmsImg + i + j*width) = maxSupp(center, p1, p2);
    }
    //fprintf(quantFile, "\n");
  }
  //fclose(quantFile);
}

__global__ 
void nms_global(float *inImg, float *nmsImg, float *gradImg, int height, int width)
{
  size_t col = blockDim.x * blockIdx.x + threadIdx.x;
  size_t row  = blockDim.y * blockIdx.y + threadIdx.y;

  float p1 = -1.0, p2 = -1.0;
  float p3 = -1.0, p4 = -1.0;
  unsigned int fAngle = 0;

  //if (col >= 0 && col < width && row >= 0 && row < height)
  if (col < width && row < height) // Since size_t is unsigned, it can't fall below 0
  {
    float angle = *(gradImg + row*width + col);

    if ((angle > -22.5 && angle <= 22.5) || (angle > 157.5) || (angle < -157.5))
      fAngle = 0;
    else if ((angle > 112.5 && angle <= 157.5) || (angle < -22.5 && angle >= -67.5))
      fAngle = 135;
    else if ((angle > 67.5 && angle <= 112.5) || (angle < -67.5 && angle >= -112.5))
      fAngle = 90;
    else if ((angle > 22.5 && angle <= 67.5) || (angle < -112.5 && angle >= -157.5))
      fAngle = 45;

    switch (fAngle)
    {
      case 0:
        p1 = getPoint(inImg, col, row+1, height, width);
        p2 = getPoint(inImg, col, row-1, height, width);
        p3 = getPoint(inImg, col, row+2, height, width);
        p4 = getPoint(inImg, col, row-2, height, width);
        break;
      case 45:
        p1 = getPoint(inImg, col-1, row-1, height, width);
        p2 = getPoint(inImg, col+1, row+1, height, width);
        p3 = getPoint(inImg, col-2, row-2, height, width);
        p4 = getPoint(inImg, col+2, row+2, height, width);
        break;
      case 90:
        p1 = getPoint(inImg, col+1, row, height, width);
        p2 = getPoint(inImg, col-1, row, height, width);
        p3 = getPoint(inImg, col+2, row, height, width);
        p4 = getPoint(inImg, col-2, row, height, width);
        break;
      case 135:
        p1 = getPoint(inImg, col+1, row-1, height, width);
        p2 = getPoint(inImg, col-1, row+1, height, width);
        p3 = getPoint(inImg, col+2, row-2, height, width);
        p4 = getPoint(inImg, col-2, row+2, height, width);
        break;
      default:
        break;
    }

    float center = getPoint(inImg, col, row, height, width);
    //*(nmsImg + i + j*width) = maxSupp(center, p1, p2, p3, p4);
    *(nmsImg + col + row*width) = maxSupp(center, p1, p2);
  }
}

