#include "hip/hip_runtime.h"
#include <stdio.h>
#include <wb.h>
#include "Otsus_Method.h"
#include "filters.h"
#include "non_max_supp.h"
#include "Edge_Connection.h"

// Use for bypassing phases for testing and debug printing
//#include "test-code.h"

#define wbCheck(stmt)                                                      \
    do                                                                     \
    {                                                                      \
        hipError_t err = stmt;                                            \
        if (err != hipSuccess)                                            \
        {                                                                  \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                    \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err)); \
            hipDeviceReset();                                             \
            return -1;                                                     \
        }                                                                  \
    } while (0)

int main(int argc, char *argv[])
{
  hipFree(0);

    //////////////////////////////
    // Parameter Initialization //
    //////////////////////////////

    hipFree(0);

    // Image parameters for wbLib
    wbArg_t args;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    float stDev;
	float stDevSq;
    size_t filterSize;
    char *inputImageFile;
    wbImage_t inputImage;
    wbImage_t outputImage;

    // Host side parameters
    float *hostInputImageData;
    float *hostGrayImageData;
    float *hostBlurImageData;
    float *hostGradMagData;
    float *hostGradPhaseData;
    float *hostNmsImageData;
    float *hostEdgeData;
    float *hostWeakEdgeData;
    float *hostThresh;

    // Device side parameters
    float *deviceInputImageData;
    float *deviceGrayImageData;
    float *deviceBlurImageData;
    float *deviceBlurTempImageData;
    float *deviceGradMagData;
    float *deviceGradPhaseData;
    float *deviceNmsImageData;
    float *deviceEdgeData;
    float *deviceWeakEdgeData;
    float *deviceThresh;

    // Otsu's Method parameters
    unsigned int *hostHistogram;
    unsigned int *deviceHistogram;


    ////////////////////
    // Image Handling //
    ////////////////////


    // Parse the input arguments
    args = wbArg_read(argc, argv);

    // Read input file
    inputImageFile = wbArg_getInputFile(args, 0);
    stDev = wbArg_getInputStdev(args);

    // Import input image
    inputImage = wbImport(inputImageFile);

    // Scrape info from input image
    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);

    // Define output image data
    hostInputImageData = wbImage_getData(inputImage);

    // Initialize memory for the output image
    // Note - input image is 3 channels. Other phases only have 1 channel
    float *outData = (float *)calloc(imageHeight * imageWidth, sizeof(float));
    outputImage = wbImage_new(imageWidth, imageHeight, 1, outData);


    ////////////////////////////////
    // Host Memory Initialization //
    ////////////////////////////////


    // Start total program timer
    wbTime_start(GPU, "Doing Computation (memory + compute)");

    // Start memory allocation timer
    wbTime_start(GPU, "Doing memory allocation");

    // Allocate memory on host
    hostGrayImageData = (float *)malloc(imageHeight * imageWidth * sizeof(float));
    hostBlurImageData = (float *)malloc(imageHeight * imageWidth * sizeof(float));
    hostGradMagData = (float *)malloc(imageHeight * imageWidth * sizeof(float));
    hostGradPhaseData = (float *)malloc(imageHeight * imageWidth * sizeof(float));
    hostNmsImageData = (float *)malloc(imageHeight * imageWidth * sizeof(float));
    hostEdgeData = (float *)malloc(imageHeight * imageWidth * sizeof(float));
    hostWeakEdgeData = (float *)malloc(imageHeight * imageWidth * sizeof(float));

    // Allocate memory on host
    hostHistogram = (unsigned int *)malloc(256 * sizeof(unsigned int));
	hostThresh = (float *)malloc(sizeof(float));


	/////////////////////////
    // Image Preprocessing //
    /////////////////////////


	// Calculate the filter size
    filterSize = ceil(stDev * 6);
	filterSize = (filterSize % 2 == 0) ? filterSize + 1 : filterSize;

	// Calculate the filter variance
	stDevSq = stDev * stDev;

    // Create filter skeleton
    double *filter = (double *)calloc(filterSize * filterSize, sizeof(double));
    double *deviceFilter;
    populate_blur_filter(filter, filterSize, stDevSq);


    //////////////////////////////////
    // Device Memory Initialization //
    //////////////////////////////////


    // Allocate memory on device
    wbCheck(hipMalloc((void **)&deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceGrayImageData, imageWidth * imageHeight * sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceBlurImageData, imageWidth * imageHeight * sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceBlurTempImageData, imageWidth * imageHeight * sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceGradMagData, imageWidth * imageHeight * sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceGradPhaseData, imageWidth * imageHeight * sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceNmsImageData, imageWidth * imageHeight * sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceEdgeData, imageWidth * imageHeight * sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceWeakEdgeData, imageWidth * imageHeight * sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceHistogram, 256 * sizeof(unsigned int)));
    wbCheck(hipMalloc((void **)&deviceThresh, sizeof(float)));
    wbCheck(hipMalloc((void **)&deviceFilter, filterSize * filterSize * sizeof(double)));

    // Initialize cuda memory
    wbCheck(hipMemset(deviceHistogram, 0, 256 * sizeof(unsigned int)));
    wbCheck(hipMemset(deviceWeakEdgeData, 0, imageWidth * imageHeight * sizeof(float)));
    wbCheck(hipMemset(deviceEdgeData, 0, imageWidth * imageHeight * sizeof(float)));

    // Stop memory allocation timer
    wbTime_stop(GPU, "Doing memory allocation");

    // Start memory copy timer
    wbTime_start(Copy, "Copying data to the GPU");

    // Copy Gaussian filter from host to device
    wbCheck(hipMemcpy(deviceFilter, filter, filterSize * filterSize * sizeof(double), hipMemcpyHostToDevice));

    // Copy input image from host to device
    wbCheck(hipMemcpy(deviceInputImageData, hostInputImageData, imageChannels * imageWidth * imageHeight * sizeof(float), hipMemcpyHostToDevice));
    wbTime_stop(Copy, "Copying data to the GPU");

   
    ///////////////////
    // GPU Execution //
    ///////////////////


    // Start computation timer
    wbTime_start(Compute, "Doing the computation on the GPU");

    // Number of threads/block is 16
    int blocksize = 16;

    // Initialize x and y block dimension to blocksize
    dim3 BlockDim(blocksize, blocksize);
    dim3 histBlockDim(1024);

    // Set x and y grid dimension
    dim3 GridDim(((imageWidth + BlockDim.x - 1) / BlockDim.x), ((imageHeight + BlockDim.y - 1) / BlockDim.y));
    dim3 histGridDim((imageWidth * imageHeight + histBlockDim.x - 1) / histBlockDim.x);
    dim3 GridDiff(((imageWidth + 14 - 1) / 14), ((imageHeight + 14 - 1) / 14));

    // Call RGB to grayscale conversion kernel
    wbTime_start(Compute, "ColorToGrayscale computation");
    	ColorToGrayscale<<<GridDim, BlockDim>>>(deviceInputImageData, deviceGrayImageData, imageWidth, imageHeight);
    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "ColorToGrayscale computation");

    // Call image burring kernel
    wbTime_start(Compute, "Conv2D computation");
         Conv2DOptRow<<<GridDiff, BlockDim>>>(deviceGrayImageData, deviceBlurTempImageData, deviceFilter, imageWidth, imageHeight, filterSize);
         Conv2DOptCol<<<GridDiff, BlockDim>>>(deviceBlurTempImageData, deviceBlurImageData, deviceFilter, imageWidth, imageHeight, filterSize);
    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Conv2D computation");

    // Call sobel filtering kernel
    wbTime_start(Compute, "GradientSobelS computation");
  		GradientSobelOpt<<<GridDim, BlockDim>>>(deviceBlurImageData, deviceGradMagData, deviceGradPhaseData, imageHeight, imageWidth); 
  	wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "GradientSobelS computation");

    // Suppress non-maximum pixels along gradient
    wbTime_start(Compute, "Non-maximum Suppression computation");
    	nms_global<<<GridDim, BlockDim>>>(deviceGradMagData, deviceNmsImageData, deviceGradPhaseData, imageHeight, imageWidth);
    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Non-maximum Suppression computation");

    // Calculate histogram of nms image
    wbTime_start(Compute, "Histogram computation");
    	OptimizedHistogramReplication<<<histGridDim, histBlockDim>>>(deviceNmsImageData, deviceHistogram, imageWidth, imageHeight);
    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Histogram computation");

    // Copy histogram to host to calculate threshold
    hipMemcpy(hostHistogram, deviceHistogram, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Calculate threshold using Otsu's Method
    wbTime_start(Compute, "Otsu's computation");
        hostThresh[0] = Otsu_Sequential_Optimized(hostHistogram, imageWidth, imageHeight);
    wbTime_stop(Compute, "Otsu's computation");

    // Copy threshold to device
    hipMemcpy(deviceThresh, hostThresh, sizeof(float), hipMemcpyHostToDevice);
    wbCheck(hipDeviceSynchronize());

    // Threshold detection shared memory kernal
    wbTime_start(Compute, "Threshold Detection computation");
    	thresh_detection_shared<<<GridDim, BlockDim>>>(deviceNmsImageData, deviceWeakEdgeData, deviceEdgeData, deviceThresh, imageWidth, imageHeight);
    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Threshold Detection computation");

    // Global Memory edge connection kernal
    wbTime_start(Compute, "Edge connection computation");
    	edge_connection_global<<<GridDim, BlockDim>>>(deviceWeakEdgeData, deviceEdgeData, imageWidth, imageHeight);
    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Edge connection computation");

    // Stop computation timer
    wbTime_stop(Compute, "Doing the computation");


    ////////////////////
    // Device Results //
    ////////////////////


    // Start device memory copy timer
    wbTime_start(Copy, "Copying data from the GPU");

    // Copy data from device back to host
    hipMemcpy(hostEdgeData, deviceEdgeData, imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);

    // Stop memory timer
    wbTime_stop(Copy, "Copying data from the GPU");

    // Stop total program timer
    wbTime_stop(GPU, "Doing Computation (memory + compute)");

    // Copy data from device back to host. Only time the first Memcpy because these are just for debug
    hipMemcpy(hostGrayImageData, deviceGrayImageData, imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostBlurImageData, deviceBlurImageData, imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostGradMagData, deviceGradMagData, imageHeight * imageWidth * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostGradPhaseData, deviceGradPhaseData, imageHeight * imageWidth * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostNmsImageData, deviceNmsImageData, imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostWeakEdgeData, deviceWeakEdgeData, imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);

    ////////////////////////
    // Logging and Output //
    ////////////////////////


    // Copy image data for output image (choose 1 - can only log one at a time for now
    // For GPU execution
    // memcpy(outData, hostGrayImageData, imageHeight*imageWidth*sizeof(float));
    // memcpy(outData, hostBlurImageData, imageHeight*imageWidth*sizeof(float));
    // memcpy(outData, hostGradMagData, imageHeight*imageWidth*sizeof(float));
    // memcpy(outData, hostGradPhaseData, imageHeight*imageWidth*sizeof(float));
    // memcpy(outData, hostNmsImageData, imageHeight*imageWidth*sizeof(float));
    // memcpy(outData, hostWeakEdgeData, imageHeight*imageWidth*sizeof(float));
       memcpy(outData, hostEdgeData, imageHeight * imageWidth * sizeof(float));

    // Export image
    char *oFile = wbArg_getOutputFile(args);
    wbExport(oFile, outputImage);


    ////////////////////
    // Debugging Info //
    ////////////////////

    // Uncomment #include test_code.h for debug statements
    #if (PRINT_DEBUG)

        // Print info
        printf("\n");
        printf("Width = %u\n", imageWidth);
        printf("Height = %u\n", imageHeight);
        printf("InputImage[0] = %f\n", hostInputImageData[0]);
        printf("Host Histogram[0] = %u\n", hostHistogram[0]);
        printf("Host Histogram[1] = %u\n", hostHistogram[1]);
        printf("Host Histogram[20] = %u\n", hostHistogram[20]);
        printf("Host Histogram[49] = %u\n", hostHistogram[49]);
        printf("Host Histogram[56] = %u\n", hostHistogram[56]);
        printf("Host Histogram[255] = %u\n", hostHistogram[255]);
        printf("Blurred Image[0] = %f\n", hostBlurImageData[0]);
        printf("Blurred Image[1] = %f\n", hostBlurImageData[1]);
        printf("Blurred Image[36] = %f\n", hostBlurImageData[36]);
        printf("Blurred Image[400] = %f\n", hostBlurImageData[400]);
        printf("Blurred Image[900] = %f\n", hostBlurImageData[900]);
        printf("Blurred Image[1405] = %f\n", hostBlurImageData[1405]);
        printf("Blurred Image[85000] = %f\n", hostBlurImageData[85000]);

        for (size_t row = 0; row < filterSize; ++row)
        {
            printf("Row=%ld of Gaussian filter = ", row);
            for (size_t col = 0; col < filterSize; ++col)
            {
                printf("%f ", filter[col + filterSize * row]);
            }
            printf("\n");
        }

        // printf("Blurred Image[0] = %f\n",hostBlurImageData[0]*255);
        // printf("Blurred [25] = %f\n", hostBlurImageData[25]*255);
        // printf("Blurred Image[290] = %f\n",hostBlurImageData[290]*255);
        // printf("Gradient magnitude at [0] = %f\n",hostGradMagData[0]);
        // printf("Gradient magnitude at [20] = %f\n",hostGradMagData[20]);
        // printf("Gradient magnitude at [9000] = %f\n",hostGradMagData[9000]);
        // printf("Gradient phase at [0] = %f\n",hostGradPhaseData[0]);
        // printf("Gradient phase at [20] = %f\n",hostGradPhaseData[20]);
        // printf("Gradient phase at [290] = %f\n",hostGradPhaseData[290]);
        // printf("NMS at [0] = %f\n",hostNmsImageData[0]);
        // printf("NMS at [20] = %f\n",hostNmsImageData[20]);
        // printf("NMS at [130] = %f\n",hostNmsImageData[130]);
        // printf("NMS at [131] = %f\n",hostNmsImageData[131]);
        printf("CUDA Otsu's Threshold = %f\n", hostThresh[0]);
        // printf("\n");
    #endif

    //////////////
    // Clean Up //
    //////////////

    // Destory all cuda memory
    hipFree(deviceInputImageData);
    hipFree(deviceGrayImageData);
    hipFree(deviceBlurImageData);
    hipFree(deviceBlurTempImageData);
    hipFree(deviceGradMagData);
    hipFree(deviceGradPhaseData);
    hipFree(deviceEdgeData);
    hipFree(deviceWeakEdgeData);
    hipFree(deviceHistogram);
    hipFree(deviceFilter);
    hipFree(deviceThresh);

    // Destroy host memory
    free(hostBlurImageData);
    free(hostGradMagData);
    free(hostGradPhaseData);
    free(hostNmsImageData);
    free(hostEdgeData);
    free(hostWeakEdgeData);
    free(hostHistogram);
    free(hostThresh);
    free(filter);

    // Destroy images
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    // Reset CUDA devices
    hipDeviceReset();

    return 0;
}